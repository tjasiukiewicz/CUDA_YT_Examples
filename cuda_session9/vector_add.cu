#include <hip/hip_runtime.h>
#include <cstdio>
#include <cassert>
#include <algorithm>
#include <numeric>

// Add 2 vector values...
template<typename T>
__global__ void vector_add(
	T * const result, const T * const source1, const T * const source2) {

	auto const idx = blockIdx.x * blockDim.x + threadIdx.x;

	result[idx] = source1[idx] + source2[idx];
}

// Verify result on CPU
template<typename T>
void verify_add(const T * const result,
	const T * const source1, const T * const source2, const size_t size) {

	printf("Vector operation verified: ");

	for (auto i = 0UL; i < size; ++i) {
		if (result[i] != (source1[i] + source2[i])) {
			printf("---->BAD on index %lu!!!\n", i);
			return;
		}
	}

	printf("OK\n");
}

int main() {
	using DataType_t = int;

	constexpr static size_t NumStreams = 8;

	constexpr static size_t Elements = 57344 * 1024;
	constexpr static size_t MemElements = Elements * sizeof(DataType_t);

	constexpr static size_t ElementsChunk = Elements / NumStreams;
	constexpr static size_t MemChunk = MemElements / NumStreams;

	constexpr static size_t ThreadsInBlock = 1024;
	constexpr static size_t Block = (Elements + ThreadsInBlock - 1) / ThreadsInBlock;

	constexpr static size_t BlockChunk = Block / NumStreams;

	hipStream_t streams[NumStreams];

	for (auto i = 0U; i < NumStreams; ++i) {
		hipStreamCreate(&streams[i]);
	}

	DataType_t * hSource1 = nullptr;
	DataType_t * hSource2 = nullptr;
	DataType_t * hResult = nullptr;

	hipHostMalloc(&hSource1, MemElements, hipHostMallocDefault);
	hipHostMalloc(&hSource2, MemElements, hipHostMallocDefault);
	hipHostMalloc(&hResult, MemElements, hipHostMallocDefault);

	std::iota(hSource1, hSource1 + Elements, 1);
	std::iota(hSource2, hSource2 + Elements, -8192);

	DataType_t * dSource1 = nullptr;
	DataType_t * dSource2 = nullptr;
	DataType_t * dResult = nullptr;

	hipMalloc(&dSource1, MemElements);
	hipMalloc(&dSource2, MemElements);
	hipMalloc(&dResult, MemElements);

	// Split by streams

	for (auto i = 0U; i < NumStreams; ++i) {
		size_t elementsOffset = ElementsChunk * i;

		hipMemcpyAsync(&dSource1[elementsOffset], &hSource1[elementsOffset],
				MemChunk, hipMemcpyHostToDevice, streams[i]);
		hipMemcpyAsync(&dSource2[elementsOffset], &hSource2[elementsOffset],
				MemChunk, hipMemcpyHostToDevice, streams[i]);

		vector_add<<<BlockChunk, ThreadsInBlock, 0, streams[i]>>>
			(&dResult[elementsOffset], &dSource1[elementsOffset], &dSource2[elementsOffset]);

		hipMemcpyAsync(&hResult[elementsOffset], &dResult[elementsOffset],
				MemChunk, hipMemcpyDeviceToHost, streams[i]);
	}

	// End split by streams

	for (auto i = 0U; i < NumStreams; ++i) {
		hipStreamDestroy(streams[i]);
	}

	hipFree(dResult);
	hipFree(dSource2);
	hipFree(dSource1);

	hipDeviceSynchronize();

	verify_add(hResult, hSource1, hSource2, Elements);

	hipHostFree(hResult);
	hipHostFree(hSource2);
	hipHostFree(hSource1);

}
