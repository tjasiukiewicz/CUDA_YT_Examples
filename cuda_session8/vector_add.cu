#include <hip/hip_runtime.h>
#include <cstdio>
#include <cassert>
#include <algorithm>
#include <numeric>

// Add 2 vector values...
template<typename T>
__global__ void vector_add(
	T * const result, const T * const source1, const T * const source2) {

	auto const idx = blockIdx.x * blockDim.x + threadIdx.x;

	result[idx] = source1[idx] + source2[idx];
}

// Verify result on CPU
template<typename T>
void verify_add(const T * const result,
	const T * const source1, const T * const source2, const size_t size) {

	printf("Vector operation verified: ");

	for (auto i = 0UL; i < size; ++i) {
		if (result[i] != (source1[i] + source2[i])) {
			printf("---->BAD on index %lu!!!\n", i);
			return;
		}
	}

	printf("OK\n");
}

int main() {
	using DataType_t = int;

	constexpr static size_t Elements = 57344 * 1024;
	constexpr static size_t MemElements = Elements * sizeof(DataType_t);
	constexpr static size_t ThreadsInBlock = 1024;
	constexpr static size_t Block = (Elements + ThreadsInBlock - 1) / ThreadsInBlock;

	DataType_t * hSource1 = nullptr;
	DataType_t * hSource2 = nullptr;
	DataType_t * hResult = nullptr;

	hipHostMalloc(&hSource1, MemElements, hipHostMallocDefault);
	hipHostMalloc(&hSource2, MemElements, hipHostMallocDefault);
	hipHostMalloc(&hResult, MemElements, hipHostMallocDefault);

	std::iota(hSource1, hSource1 + Elements, 1);
	std::iota(hSource2, hSource2 + Elements, -8192);

	DataType_t * dSource1 = nullptr;
	DataType_t * dSource2 = nullptr;
	DataType_t * dResult = nullptr;

	hipMalloc(&dSource1, MemElements);
	hipMalloc(&dSource2, MemElements);
	hipMalloc(&dResult, MemElements);

	hipMemcpyAsync(dSource1, hSource1, MemElements, hipMemcpyHostToDevice);
	hipMemcpyAsync(dSource2, hSource2, MemElements, hipMemcpyHostToDevice);

	vector_add<<<Block, ThreadsInBlock>>>(dResult, dSource1, dSource2);

	hipMemcpyAsync(hResult, dResult, MemElements, hipMemcpyDeviceToHost);

	hipFree(dResult);
	hipFree(dSource2);
	hipFree(dSource1);

	hipDeviceSynchronize();

	verify_add(hResult, hSource1, hSource2, Elements);

	hipHostFree(hResult);
	hipHostFree(hSource2);
	hipHostFree(hSource1);

}
