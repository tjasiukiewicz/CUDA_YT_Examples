#include <hip/hip_runtime.h>
#include <cstdio>
#include <algorithm>
#include <numeric>

// __host__   - dane są dostępne w ramach operacji na CPU
// __device__ - dane są dostępne w ramach operacji na GPU

template<typename T, typename Func>
__global__ void map(T * const destination, const T * const source, const size_t size, Func func) {
	auto const idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size) {
		destination[idx] = func(source[idx]);
	}
}

template<typename T>
void showData(const T * const data, const size_t size) {
	// Dla prostoty przykładu, pozostawiam jedynie format %d dla obsługiwanych danych
	std::for_each(data, data + size, [](T a) { printf("%3d ", a); });
	putchar('\n');
}

int main() {
	using DataType_t = int;

	constexpr static size_t Elements = 11;
	constexpr static size_t MemElements = Elements * sizeof(DataType_t);
	constexpr static size_t ThreadsInBlock = 4;
	constexpr static size_t Block = (Elements + ThreadsInBlock - 1) / ThreadsInBlock;

	auto hSource = static_cast<DataType_t *>(malloc(MemElements));
	auto hDestination = static_cast<DataType_t *>(malloc(MemElements));

	DataType_t * dSource = nullptr;
	DataType_t * dDestination = nullptr;

	hipMalloc(&dSource, MemElements);
	hipMalloc(&dDestination, MemElements);

	std::iota(hSource, hSource + Elements, 0);

	showData(hSource, Elements);

	hipMemcpy(dSource, hSource, MemElements, hipMemcpyHostToDevice);

	map<<<Block, ThreadsInBlock>>>(dDestination, dSource, Elements, [] __device__ (auto a) { return 10 * a; });

	hipMemcpy(hDestination, dDestination, MemElements, hipMemcpyDeviceToHost);

	showData(hDestination, Elements);

	hipFree(dDestination);
	hipFree(dSource);

	free(hDestination);
	free(hSource);

	hipDeviceSynchronize();
}
