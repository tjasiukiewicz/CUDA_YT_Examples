#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cstdint>
#include <cstddef>
#include <chrono>
#include <iostream>
#include "helper_cuda.cuh"

using DataType_t = float;

constexpr static size_t Length = 33554432; // 32 MB elements

constexpr static size_t ThreadsInBlock = 1024;

// Kernel
// Fake, compute intensive kernel...
template<typename T>
__global__ void calculate(
		const T * const x, const T * const y,
		T * const distance, T * const sine,
		const std::size_t length
) {

	auto const tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < length) {
		auto & mx = x[tid];
		auto & my = y[tid];
		auto & md = distance[tid];
		md = sqrtf(mx * mx + my * my);
	}
}

int main() {

	constexpr static size_t Elements = Length;
	constexpr static size_t MemElements = Elements * sizeof(DataType_t);
	constexpr static size_t Blocks = (Elements + ThreadsInBlock - 1) / ThreadsInBlock;

	// Start for CPU
	auto hStart = std::chrono::high_resolution_clock::now();

	DataType_t * hX, * hY, * hDistance, * hSine;
	hX = hY = hDistance = hSine = nullptr;

	cudaCheckError(hipHostAlloc(&hX, MemElements, hipHostMallocWriteCombined));
	cudaCheckError(hipHostAlloc(&hY, MemElements, hipHostMallocWriteCombined));
	cudaCheckError(hipHostAlloc(&hDistance, MemElements, hipHostRegisterPortable));
	cudaCheckError(hipHostAlloc(&hSine, MemElements, hipHostRegisterPortable));

	// Fill X, Y coord
	/*
	srand(0);
	for (auto i = 0UL; i < Elements; ++i) {
		// [-1, 1]
		hX[i] = ((rand() * 2.0F) / RAND_MAX) - 1.0F;
		hY[i] = ((rand() * 2.0F) / RAND_MAX) - 1.0F;
	}
	*/

	// Start for GPU
	auto dStart = std::chrono::high_resolution_clock::now();

	DataType_t * dX, * dY, * dDistance, * dSine;
	dX = dY = dDistance = dSine = nullptr;

	for (auto ptr: {&dX, &dY, &dDistance, &dSine}) {
		cudaCheckError(hipMalloc(ptr, MemElements));
	}

	cudaCheckLastErrorCont();
	hipMemcpyAsync(dX, hX, MemElements, hipMemcpyHostToDevice);
	cudaCheckLastError();
	hipMemcpyAsync(dY, hY, MemElements, hipMemcpyHostToDevice);
	cudaCheckLastError();

	cudaCheckLastErrorCont();
	calculate<<<Blocks, ThreadsInBlock>>>(dX, dY, dDistance, dSine, Elements);
	cudaCheckLastError();

	cudaCheckLastErrorCont();
	hipMemcpyAsync(hDistance, dDistance, MemElements, hipMemcpyDeviceToHost);
	cudaCheckLastError();
	hipMemcpyAsync(hSine, dSine, MemElements, hipMemcpyDeviceToHost);
	cudaCheckLastError();

	for (auto ptr: {dX, dY, dDistance, dSine}) {
		cudaCheckError(hipFree(ptr));
	}

	// End for GPU
	auto dStop = std::chrono::high_resolution_clock::now();

	for (auto ptr: {hX, hY, hDistance, hSine}) {
		cudaCheckError(hipHostFree(ptr));
	}

	// End for CPU
	auto hStop = std::chrono::high_resolution_clock::now();

	std::cout << "GPU Execution time: "
		<< std::chrono::duration_cast<std::chrono::milliseconds>(dStop - dStart).count()
		<< " ms.\n";

	std::cout << "CPU Execution time: "
		<< std::chrono::duration_cast<std::chrono::milliseconds>(hStop - hStart).count()
		<< " ms.\n";
}
